// matmul_cublas.cu
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <cstdlib>

#define CHECK_CUDA(call)                                                     \
    do {                                                                     \
        hipError_t err = (call);                                            \
        if (err != hipSuccess) {                                            \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__     \
                      << " code=" << err << " \"" << hipGetErrorString(err) \
                      << "\"" << std::endl;                                  \
            std::exit(EXIT_FAILURE);                                         \
        }                                                                    \
    } while (0)

#define CHECK_CUBLAS(call)                                                   \
    do {                                                                     \
        hipblasStatus_t stat = (call);                                        \
        if (stat != HIPBLAS_STATUS_SUCCESS) {                                 \
            std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__   \
                      << " code=" << stat << std::endl;                      \
            std::exit(EXIT_FAILURE);                                         \
        }                                                                    \
    } while (0)

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <device> <matrix_size>\n";
        return EXIT_FAILURE;
    }

    int dev = std::atoi(argv[1]);
    int N   = std::atoi(argv[2]);

    // Select GPU
    CHECK_CUDA(hipSetDevice(dev));

    // cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Allocate device matrices A, B, C (uninitialized, like cupy.empty)
    size_t bytes = size_t(N) * N * sizeof(float);
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, bytes));
    CHECK_CUDA(hipMalloc(&d_B, bytes));
    CHECK_CUDA(hipMalloc(&d_C, bytes));

    // Warm-up: one matmul to initialize kernels / cache
    const float alpha = 1.0f, beta = 0.0f;
    CHECK_CUBLAS(
        hipblasSgemm(handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, N, N,
                    &alpha,
                    d_A, N,
                    d_B, N,
                    &beta,
                    d_C, N)
    );
    // Ensure warm-up completed
    CHECK_CUDA(hipDeviceSynchronize());

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record start, run 20 matmuls, record stop
    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < 20; ++i) {
        CHECK_CUBLAS(
            hipblasSgemm(handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        N, N, N,
                        &alpha,
                        d_A, N,
                        d_B, N,
                        &beta,
                        d_C, N)
        );
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));

    std::cout << "Matrix size: " << N << "×" << N << "\n"
              << "Total time for 20 calls: " << ms << " ms\n"
              << "Average per call: " << (ms / 20.0f) << " ms\n";

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}
