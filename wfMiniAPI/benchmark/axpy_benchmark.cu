#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(err));                                 \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

#define CHECK_CUBLAS(call)                                                    \
    do {                                                                      \
        hipblasStatus_t st = call;                                             \
        if (st != HIPBLAS_STATUS_SUCCESS) {                                    \
            fprintf(stderr, "cuBLAS Error %s:%d: %d\n", __FILE__, __LINE__, st); \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

int main() {
    const int N = 1024 * 1024;
    const float alpha = 2.5f;
    const int n_warmup = 3;
    const int n_repeat = 50;

    // Host allocations
    float *h_x = (float*)malloc(N * sizeof(float));
    float *h_y = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; ++i) {
        h_x[i] = 1.0f;      // or whatever initialization
        h_y[i] = 0.0f;
    }

    // Device allocations
    float *d_x, *d_y;
    CHECK_CUDA(hipMalloc((void**)&d_x, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_y, N * sizeof(float)));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Timing events
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Warm‐up
    for (int i = 0; i < n_warmup; ++i) {
        CHECK_CUBLAS(hipblasSaxpy(handle, N, &alpha, d_x, 1, d_y, 1));
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // Timed repeats
    float total_ms = 0.0f;
    for (int i = 0; i < n_repeat; ++i) {
        CHECK_CUDA(hipEventRecord(start, 0));
        CHECK_CUBLAS(hipblasSaxpy(handle, N, &alpha, d_x, 1, d_y, 1));
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(stop));
        float iter_ms = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&iter_ms, start, stop));
        total_ms += iter_ms;
    }

    float avg_ms = total_ms / n_repeat;
    double bytes = double(N) * sizeof(float) * 2;  // read x and read/write y
    double bandwidth = bytes / (avg_ms / 1e3) / 1e9;  // GB/s

    printf("AXPY (N=%d) average over %d runs: %f ms, bandwidth = %f GB/s\n",
           N, n_repeat, avg_ms, bandwidth);

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_x);
    hipFree(d_y);
    free(h_x);
    free(h_y);

    return 0;
}

