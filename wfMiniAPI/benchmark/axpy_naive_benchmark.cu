#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(err));                                 \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

__global__ void axpy_kernel(int N, float alpha, const float* x, float* y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        y[idx] += alpha * x[idx];
    }
}

int main() {
    const int N = 1024 * 1024 * 32;
    const float alpha = 1.1f;
    const int n_warmup = 3;
    const int n_repeat = 50;

    float *h_x = (float*)malloc(N * sizeof(float));
    float *h_y = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; ++i) {
        h_x[i] = 1.0f;
        h_y[i] = 0.0f;
    }

    float *d_x, *d_y;
    CHECK_CUDA(hipMalloc((void**)&d_x, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_y, N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice));

    const int TPB = 256;
    int blocks = (N + TPB - 1) / TPB;

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    for (int i = 0; i < n_warmup; ++i) {
        axpy_kernel<<<blocks, TPB>>>(N, alpha, d_x, d_y);
    }
    CHECK_CUDA(hipDeviceSynchronize());

    float total_ms = 0.0f;
    for (int i = 0; i < n_repeat; ++i) {
        CHECK_CUDA(hipEventRecord(start, 0));
        axpy_kernel<<<blocks, TPB>>>(N, alpha, d_x, d_y);
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(stop));
        float iter_ms = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&iter_ms, start, stop));
        total_ms += iter_ms;
    }

    float avg_ms = total_ms / n_repeat;
    printf("AXPY kernel (N=%d) average over %d runs: %f ms\n",
           N, n_repeat, avg_ms);

    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));
    free(h_x);
    free(h_y);

    return 0;
}

