#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(err));                                 \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

#define CHECK_CUFFT(call)                                                     \
    do {                                                                      \
        hipfftResult err = call;                                               \
        if (err != HIPFFT_SUCCESS) {                                           \
            fprintf(stderr, "cuFFT Error %s:%d: %d\n", __FILE__, __LINE__, err); \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

int main() {
    const int N         = 1024;            // length of each FFT
    const int batch     = 1024;            // number of transforms (one per column)
    const int n_warmup  = 3;
    const int n_repeat  = 50;

    // host-side real input (simulate data_in); we'll pack into complex
    size_t real_elems = size_t(N) * batch;
    float *h_real = (float*)malloc(real_elems * sizeof(float));
    for (size_t i = 0; i < real_elems; ++i) {
        h_real[i] = 1.0f;  // or load your actual data here
    }

    // Device buffers: complex in/out
    hipfftComplex *d_in = nullptr, *d_out = nullptr;
    size_t complex_bytes = real_elems * sizeof(hipfftComplex);
    CHECK_CUDA(hipMalloc(&d_in,  complex_bytes));
    CHECK_CUDA(hipMalloc(&d_out, complex_bytes));

    // Pack real→complex on host and copy once
    hipfftComplex *h_pack = (hipfftComplex*)malloc(complex_bytes);
    for (size_t i = 0; i < real_elems; ++i) {
        h_pack[i].x = h_real[i];
        h_pack[i].y = 0.0f;
    }
    CHECK_CUDA(hipMemcpy(d_in, h_pack, complex_bytes, hipMemcpyHostToDevice));
    free(h_pack);
    free(h_real);

    // Create a batched 1D C2C plan
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlan1d(&plan, N, HIPFFT_C2C, batch));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Warm‑up (not timed)
    for (int i = 0; i < n_warmup; ++i) {
        CHECK_CUFFT(hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD));
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // Timed repeats
    float total_ms = 0.0f;
    for (int i = 0; i < n_repeat; ++i) {
        CHECK_CUDA(hipEventRecord(start, 0));
        CHECK_CUFFT(hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD));
        CHECK_CUDA(hipEventRecord(stop,  0));
        CHECK_CUDA(hipEventSynchronize(stop));

        float iter_ms = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&iter_ms, start, stop));
        total_ms += iter_ms;
    }

    float avg_ms = total_ms / n_repeat;

    // Approximate FLOPS: ~5·N·log2(N) per transform
    double flops_per_transform = 5.0 * N * std::log2(double(N));
    double total_flops         = flops_per_transform * batch;
    double gflops              = total_flops / (avg_ms/1e3) / 1e9;

    printf("cuFFT C2C 1D FFT (N=%d, batch=%d)\n", N, batch);
    printf("  Average over %d runs: %f ms\n", n_repeat, avg_ms);
    printf("  Approx. throughput: %f GFLOPS\n", gflops);

    // Cleanup
    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK_CUDA(hipFree(d_in));
    CHECK_CUDA(hipFree(d_out));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return 0;
}

