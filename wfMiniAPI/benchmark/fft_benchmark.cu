#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(err));                                 \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

#define CHECK_CUFFT(call)                                                     \
    do {                                                                      \
        hipfftResult err = call;                                               \
        if (err != HIPFFT_SUCCESS) {                                           \
            fprintf(stderr, "cuFFT Error %s:%d: %d\n", __FILE__, __LINE__, err); \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

int main() {
    const int N         = 1024;            
    const int batch     = 1024;            
    const int n_warmup  = 3;
    const int n_repeat  = 50;

    size_t real_elems = size_t(N) * batch;
    float *h_real = (float*)malloc(real_elems * sizeof(float));
    for (size_t i = 0; i < real_elems; ++i) {
        h_real[i] = 1.0f;  
    }

    hipfftComplex *d_in = nullptr, *d_out = nullptr;
    size_t complex_bytes = real_elems * sizeof(hipfftComplex);
    CHECK_CUDA(hipMalloc(&d_in,  complex_bytes));
    CHECK_CUDA(hipMalloc(&d_out, complex_bytes));

    hipfftComplex *h_pack = (hipfftComplex*)malloc(complex_bytes);
    for (size_t i = 0; i < real_elems; ++i) {
        h_pack[i].x = h_real[i];
        h_pack[i].y = 0.0f;
    }
    CHECK_CUDA(hipMemcpy(d_in, h_pack, complex_bytes, hipMemcpyHostToDevice));
    free(h_pack);
    free(h_real);

    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlan1d(&plan, N, HIPFFT_C2C, batch));

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    for (int i = 0; i < n_warmup; ++i) {
        CHECK_CUFFT(hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD));
    }
    CHECK_CUDA(hipDeviceSynchronize());

    float total_ms = 0.0f;
    for (int i = 0; i < n_repeat; ++i) {
        CHECK_CUDA(hipEventRecord(start, 0));
        CHECK_CUFFT(hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD));
        CHECK_CUDA(hipEventRecord(stop,  0));
        CHECK_CUDA(hipEventSynchronize(stop));

        float iter_ms = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&iter_ms, start, stop));
        total_ms += iter_ms;
    }

    float avg_ms = total_ms / n_repeat;

    printf("cuFFT C2C 1D FFT (N=%d, batch=%d)", N, batch);
    printf("  Average over %d runs: %f ms\n", n_repeat, avg_ms);

    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK_CUDA(hipFree(d_in));
    CHECK_CUDA(hipFree(d_out));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return 0;
}

