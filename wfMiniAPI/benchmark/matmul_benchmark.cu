#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(err));                                 \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

#define CHECK_CUBLAS(call)                                                    \
    do {                                                                      \
        hipblasStatus_t st = call;                                             \
        if (st != HIPBLAS_STATUS_SUCCESS) {                                    \
            fprintf(stderr, "cuBLAS Error %s:%d: %d\n", __FILE__, __LINE__, st); \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

int main() {
    const int N = 4096;
    const size_t bytes = size_t(N) * N * sizeof(float);
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    const int n_warmup = 3;
    const int n_repeat = 50;

    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    for (int i = 0; i < N*N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
        h_C[i] = 0.0f;
    }

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, bytes));
    CHECK_CUDA(hipMalloc(&d_B, bytes));
    CHECK_CUDA(hipMalloc(&d_C, bytes));

    CHECK_CUDA(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C, h_C, bytes, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    for (int i = 0; i < n_warmup; ++i) {
        CHECK_CUBLAS(hipblasSgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, N, N,
            &alpha,
            d_A, N,
            d_B, N,
            &beta,
            d_C, N
        ));
    }
    CHECK_CUDA(hipDeviceSynchronize());

    float total_ms = 0.0f;
    for (int i = 0; i < n_repeat; ++i) {
        CHECK_CUDA(hipEventRecord(start, 0));
        CHECK_CUBLAS(hipblasSgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, N, N,
            &alpha,
            d_A, N,
            d_B, N,
            &beta,
            d_C, N
        ));
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(stop));

        float iter_ms = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&iter_ms, start, stop));
        total_ms += iter_ms;
    }

    float avg_ms = total_ms / n_repeat;

    printf("cuBLAS SGEMM (N=%d) average over %d runs: %f ms\n",
           N, n_repeat, avg_ms);

    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
