#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(err));                                 \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

#define CHECK_CUBLAS(call)                                                    \
    do {                                                                      \
        hipblasStatus_t st = call;                                             \
        if (st != HIPBLAS_STATUS_SUCCESS) {                                    \
            fprintf(stderr, "cuBLAS Error %s:%d: %d\n", __FILE__, __LINE__, st); \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

int main() {
    const int N = 1024;
    const size_t bytes = size_t(N) * N * sizeof(float);
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    const int n_warmup = 3;
    const int n_repeat = 50;

    // Host allocations & initialize
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    for (int i = 0; i < N*N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
        h_C[i] = 0.0f;
    }

    // Device allocations
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, bytes));
    CHECK_CUDA(hipMalloc(&d_B, bytes));
    CHECK_CUDA(hipMalloc(&d_C, bytes));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C, h_C, bytes, hipMemcpyHostToDevice));

    // cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Warm‑up SGEMM calls (not timed)
    for (int i = 0; i < n_warmup; ++i) {
        CHECK_CUBLAS(hipblasSgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, N, N,
            &alpha,
            d_A, N,
            d_B, N,
            &beta,
            d_C, N
        ));
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // Timed repeats
    float total_ms = 0.0f;
    for (int i = 0; i < n_repeat; ++i) {
        CHECK_CUDA(hipEventRecord(start, 0));
        CHECK_CUBLAS(hipblasSgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, N, N,
            &alpha,
            d_A, N,
            d_B, N,
            &beta,
            d_C, N
        ));
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(stop));

        float iter_ms = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&iter_ms, start, stop));
        total_ms += iter_ms;
    }

    float avg_ms = total_ms / n_repeat;
    double flops = 2.0 * double(N) * N * N;  // 2·N³ operations for SGEMM
    double gflops = (flops / (avg_ms / 1e3)) / 1e9;

    printf("cuBLAS SGEMM (N=%d) average over %d runs: %f ms → %f GFLOPS\n",
           N, n_repeat, avg_ms, gflops);

    // Cleanup
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

